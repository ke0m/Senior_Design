#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <float.h>
#include "hip/hip_complex.h"
#include "time.h"
#include "hip/hip_runtime.h"
#include <iostream>

__global__ void test(float loop, float *out)
{
    register float a=1.0f;
    register float b=1.0f;
    register float c=1.0f;
    register float d=1.0f;
    register float e=1.0f;
    register float f=1.0f;
    register float g=1.0f;
    register float h=1.0f;

    for (float x=0;x<loop;x++)
    {
        a+=x*loop;
        b+=x*loop;
        c+=x*loop;
        d+=x*loop;
        e+=x*loop;
        f+=x*loop;
        g+=x*loop;
        h+=x*loop;

        a+=x*loop;
        b+=x*loop;
        c+=x*loop;
        d+=x*loop;
        e+=x*loop;
        f+=x*loop;
        g+=x*loop;
        h+=x*loop;

        a+=x*loop;
        b+=x*loop;
        c+=x*loop;
        d+=x*loop;
        e+=x*loop;
        f+=x*loop;
        g+=x*loop;
        h+=x*loop;

        a+=x*loop;
        b+=x*loop;
        c+=x*loop;
        d+=x*loop;
        e+=x*loop;
        f+=x*loop;
        g+=x*loop;
        h+=x*loop;

        a+=x*loop;
        b+=x*loop;
        c+=x*loop;
        d+=x*loop;
        e+=x*loop;
        f+=x*loop;
        g+=x*loop;
        h+=x*loop;
    }
    if (out!=NULL) *out=a+b+c+d+e+f+g+h;
}

int main(int argc, char *argv[])
{
    float timestamp;
    hipEvent_t event_start,event_stop;
    // Initialise
    hipDeviceReset();
    hipSetDevice(0);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    // Allocate and generate buffers
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    hipEventRecord(event_start, 0);
    dim3 threadsPerBlock;
    dim3 blocks;
    threadsPerBlock.x=32;
    threadsPerBlock.y=32;
    threadsPerBlock.z=1;
    blocks.x=1;
    blocks.y=1000;
    blocks.z=1;

    test<<<blocks,threadsPerBlock,0>>>(1000,NULL);

    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Calculated in %f\n", timestamp);
	double mflops = (1000*1024*30*80*(1.0e-6))/timestamp;
	std::cout << "MegaFlops: " << mflops << std::endl;
	system("pause");
}