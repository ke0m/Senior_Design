#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include "Stopwatch.h"

#include <stdio.h>


__global__ void arraySum(const float *d_xx, float* d_yy, int dims, int iters)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if(x >= dims){return;}
	register float r_d_zz = d_xx[x] + d_yy[x];
	for(int i = 0; i < iters; i++)
		r_d_zz += d_yy[i];
	d_yy[x] = r_d_zz;
}

int main(void)
{
 
	
	
	return 0;

}