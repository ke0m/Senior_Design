#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string>
#include "Stopwatch.h"

#include <stdio.h>


__global__ void arraySum(const float *d_xx, float* d_yy, int dims, int iters)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if(x >= dims){return;}
	register float r_d_zz = d_xx[x] + d_yy[x];
	for(int i = 0; i < iters; i++)
		r_d_zz += d_yy[i];
	d_yy[x] = r_d_zz;
}

int main(void)
{
 
	 //Declaring Variables for for loops
    int minIters = 1;
    int maxIters = 4096;
    int minSize = 100;
    int maxSize = 1000;
    int a = 0;	
    int b = 0;

		//writing flops data to a file
	std::ofstream outputfile;
	outputfile.open("..\\flops_results_files\\cuda_result_biggeriters.txt");
	
	hipError_t cudaStatus;

	Stopwatch sw;

	//Outer most loop: This loop should encompass all code and is used in order to loop over
    //different values of the iterations and array size to create the data for the plot.
	//for(int grpsize = 128; grpsize < 2048; grpsize*=2){
		for(int i = minSize; i < maxSize; i+=100){
			for(int j = minIters; j < maxIters; j*=2){

            
				//Initializing the three 2D arrays
            
				int n1 = i;
				int n2 = i+1;
				int iters = j;
				const long dims = n1*n2;
            
				float **h_xx = new float*[n1];
				float **h_yy = new float*[n1];
				float **h_zz = new float*[n1];
            
				for(int x = 0; x<n1; x++){
                
					h_xx[x] = new float [n2];
					h_yy[x] = new float [n2];
					h_zz[x] = new float [n2];
                
					//Initializing the arrays.
					for(int y = 0; y<n2; y++){

						h_xx[x][y] = x+y;
						h_yy[x][y] = x+y;
                    
					}
                
				}
            
				//Declaring the three 1D arrays
				float *h_xx1 = new float[dims];
				float *h_yy1 = new float[dims];
				float *h_zz1 = new float[dims];
                
                
				//packing arrays
				int k = 0;
				for (int x = 0; x < n1; x++){
					for (int y = 0; y < n2; y++){
						h_xx1[k] = h_xx[x][y];
						h_yy1[k] = h_yy[x][y];
						k++;
					}
                    
				}

				float *d_xx, *d_yy, *d_zz;

				cudaStatus = hipMalloc((void**)&d_xx, sizeof(float) * dims);
				if(cudaStatus != hipSuccess){
					std::cout << "hipMalloc Failed!" << std::endl;
					system("pause");
					exit(1);
				}

				cudaStatus = hipMalloc((void**)&d_yy, sizeof(float) * dims);
				if(cudaStatus != hipSuccess){
					std::cout << "hipMalloc Failed!" << std::endl;
					system("pause");
					exit(1);
				}

				cudaStatus = hipMemcpy(d_xx, h_xx1, dims * sizeof(float), hipMemcpyHostToDevice);
				if(cudaStatus != hipSuccess){
					std::cout << "cudaMemCpy Failed" << std::endl;
					system("pause");
					exit(1);
				}

				cudaStatus = hipMemcpy(d_yy, h_yy1, dims * sizeof(float), hipMemcpyHostToDevice);
				if(cudaStatus != hipSuccess){
					std::cout << "cudaMemCpy Failed" << std::endl;
					system("pause");
					exit(1);
				}

				int maxTime = 5;
				int count = 0;
				sw.restart();
				while(sw.getTime() < maxTime){

					//int threadsPerBlock = grpsize;
					int threadsPerBlock = 1024;
					//dim3 blockSize(threadsPerBlock);
					//dim3 gridSize((ceil(dims/threadsPerBlock) * threadsPerBlock));
					int blocks = (dims + threadsPerBlock)/threadsPerBlock;
					arraySum<<<blocks, threadsPerBlock>>>(d_xx, d_yy, dims, j);

					cudaStatus = hipGetLastError();
					if(cudaStatus != hipSuccess){
						std::cout << "arraySum kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
						system("pause");
						exit(1);
					}
			
					count++;
			
				}


				sw.stop();

				cudaStatus = hipDeviceSynchronize();
				if(cudaStatus != hipSuccess){
					std::cout << "cudaStatus returned error code " << cudaStatus << " after launching arraySum kernel." << std::endl;
					system("pause");
					exit(1);
				}
			

				cudaStatus = hipMemcpy(h_zz1, d_yy, dims*sizeof(float), hipMemcpyDeviceToHost);
				if(cudaStatus != hipSuccess){
					std::cout << "hipMemcpy failed!" << std::endl;
					system("pause");
					exit(1);
				}


				hipFree(d_xx);
				hipFree(d_yy);



				//unpacking the 1D array
				k = 0;
				for (int x = 0; x < n1; x++){
					for (int y = 0; y < n2; y++){
						h_zz[x][y] = h_zz1[k];
						k++;
					}
				}
                
				delete [] h_xx1;
				delete [] h_yy1;
				delete [] h_zz1;
      


				for (int x = 0; x < n1; x++)
				{
					delete [] h_xx[x];
					delete [] h_yy[x];
					delete [] h_zz[x];
				}
            
				delete [] h_xx;
				delete [] h_yy;
				delete [] h_zz;
            

            
				float n1f = (float) n1;
				float n2f = (float) n2;
				float countf = (float) count;

				std::cout << "Iters: " << iters << std::endl;
				//outputfile << "Iters: " << iters << std::endl;
				std::cout << "n1: " << n1f << std::endl;
				//outputfile << "n1: " << n1f << std::endl;
				std::cout << "n2: " << n2f << std::endl;
				//outputfile << "n2: " << n2f << std::endl;
				//std::cout << "Threads per Block: " << grpsize << std::endl;
				//outputfile << "Threads per Block: " << grpsize << std::endl;
				std::cout << "count: " << countf << std::endl;
				//outputfile<< "count: " << countf << std::endl;
				std::cout << "Time: " << sw.getTime() << std::endl;
				//outputfile << "Time: " << sw.getTime() << std::endl;
            
				float mflops = n1f*n2f*countf*iters*1.0e-06/sw.getTime();
            
				//std::cout << "Number of MegaFLOPs: " << n1f*n2f*500*countf*1.0e-6 << std::endl;
				std::cout << mflops << " MegaFLOPS" << std::endl;
				//outputfile << mflops << " MegaFLOPS" << std::endl;
				std::cout << std::endl;
				//outputfile << std::endl;

				//outputfile << mflops << std::endl;


				b++;

	
			}
	
			a++;
			std::cout << std::endl;
			outputfile << std::endl;


		//}

	}

	outputfile.close();					
	
	return 0;

}